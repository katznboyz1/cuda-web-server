#include "hip/hip_runtime.h"
#include <cuda_device_runtime_api.h>
#include <iostream>
#include <boost/property_tree/json_parser.hpp>
#include <boost/property_tree/ptree.hpp>

#define BLOCK_SIZE 256

using namespace std;

const std::string CONFIG_PATH = "./config.json";

// temporary values, will be set by readConfig()
uint32_t MAX_KERNELS = 0;
uint16_t OUTGOING_PORT_ADDR = 0;
std::string WEB_ROOT_DIR = "/dev/null";

__host__
void readConfig(std::string configPath) {

    boost::property_tree::ptree loadedPTreeRoot;
    boost::property_tree::read_json(CONFIG_PATH, loadedPTreeRoot);

    MAX_KERNELS = loadedPTreeRoot.get_child("threads").get_value<uint32_t>();
    OUTGOING_PORT_ADDR = loadedPTreeRoot.get_child("port").get_value<uint16_t>();
    WEB_ROOT_DIR = loadedPTreeRoot.get_child("web_root").get_value<std::string>();
}

__global__
void webKernel() {

    const uint32_t THREAD_ID = (blockIdx.x * blockDim.x) + threadIdx.x;
}

__host__
int main() {

    // read the config file
    std::cout << "[main.cu] READING CONFIGURATION FILE" << endl;
    readConfig(CONFIG_PATH);

    // spawn the kernels
    std::cout << "[main.cu] STARTING SERVER ON PORT " << std::to_string(OUTGOING_PORT_ADDR) << " WITH " << std::to_string(MAX_KERNELS) << " THREADS USING THE WEB ROOT DIR " << WEB_ROOT_DIR << endl;
    webKernel<<<BLOCK_SIZE * ((MAX_KERNELS / BLOCK_SIZE) + 1), BLOCK_SIZE>>>();

    // handle a graceful exit
    hipDeviceSynchronize();
    std::cout << "[main.cu] EXITED GRACEFULLY";

    return 0;
}